/**
 * This is an example of a cuda program that performs a scalar product.
  * It uses the CUDA runtime API to manage memory and execute kernel functions.
  * The vector size can be changed by changing the value of NUM_ELEMENTS.
  */
#include <stdio.h>
#include "hip/driver_types.h"
#include <hip/hip_runtime.h>
#include <time.h>        // For timing

#define NUM_ELEMENTS 1024 * 1024 * 16  // Vector size, you can change it to your needs
#define BLOCK_SIZE 512                  // Block size for CUDA kernel

__global__ void dot(float* a, float* b, float *c) {

    __shared__ float temp[BLOCK_SIZE];
    int idx = threadIdx.x + blockDim.x * blockIdx.x; // This calculates the global index of the current thread.
    
    temp[threadIdx.x] = a[idx] * b[idx]; 
    __syncthreads(); // This synchronizes all threads in the block.
    
    if (threadIdx.x == 0) { // Only the first thread computes the sum
        float sum = 0.0; 
        for(int i = 0; i < BLOCK_SIZE; ++i)
            sum += temp[i];
        atomicAdd(c, sum); // This adds the sum to the value pointed by the pointer "c". The addition is performed in a thread-safe way, preventing race conditions
    }
}

int main(void) {
    float *a, *b, *c;  // host copies of vectors a, b
    float *d_a, *d_b, *d_c;  // device copies of vectors a, b, c
    
    clock_t start, end;  // variables for time measurement

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, sizeof(float) * NUM_ELEMENTS);
    hipMalloc((void **)&d_b, sizeof(float) * NUM_ELEMENTS);
    hipMalloc((void **)&d_c, sizeof(float));
    
    // Allocate space for host copies of a, b, c and setup input values
    a = (float*)malloc(sizeof(float) * NUM_ELEMENTS);
    b = (float*)malloc(sizeof(float) * NUM_ELEMENTS);
    c = (float*)malloc(sizeof(float));
    c[0] = 0.0;
    
    // Fix the seed of the PRNG to get reproducible results.
    srand(42);    
    
    // Initialize the vectors a and b with some values.
    for (int i = 0; i < NUM_ELEMENTS; ++i) {
        a[i] = 1.0;  // random numbers between [0,1]
        b[i] = (float)rand() / (float)RAND_MAX;  // random numbers between [0,1]
    }

    // To check if the dot product is correct
    float sum = 0.;
    for (int i = 0; i < NUM_ELEMENTS; ++i){
        sum += b[i];
    }
    printf("Sum of b %.6e\n", sum);
    start = clock();   // Start timing

    // Copy inputs to device
    hipMemcpy(d_a, a, sizeof(float) * NUM_ELEMENTS, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * NUM_ELEMENTS, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, sizeof(float), hipMemcpyHostToDevice);
    // Launch dot() kernel on GPU with N blocks
    int threads = BLOCK_SIZE;
    int blocks = (NUM_ELEMENTS) / threads;
    printf("Number of elements: %d; BLOCK_SIZE: %d; Number of blocks: %d\n", NUM_ELEMENTS, BLOCK_SIZE, blocks);

    // Execute the kernel
    dot<<<blocks,threads>>>(d_a, d_b, d_c);
    
    // Copy result back to host
    hipMemcpy(c, d_c, sizeof(float), hipMemcpyDeviceToHost);

    end = clock();   // End timing

    printf("Result: %.6e; diff: %.6e\n", c[0], sum - c[0]);
    printf("Time elapsed: %.2f ms\n", (double)(end - start) / CLOCKS_PER_SEC * 1000);

    // Cleanup
    free(a); free(b);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}